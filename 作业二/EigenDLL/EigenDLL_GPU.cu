#include "EigenDLL.h"
#include <cstdio>        // ���fprintf֧��
#include <cstdlib>       // ���exit֧��
#include <hip/hip_runtime.h>
#include <hipsolver.h>  // cuSOLVER��Ҫͷ�ļ�
#include <chrono>

#define CUDA_CHECK(fn) { \
    hipError_t err = (fn); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error [%s:%d]: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CUSOLVER_CHECK(fn) { \
    hipsolverStatus_t status = (fn); \
    if (status != HIPSOLVER_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSOLVER error [%s:%d]: %d\n", __FILE__, __LINE__, (int)status); \
        exit(EXIT_FAILURE); \
    } \
}

extern "C" {
    double calculate_eigenvalues_gpu(double* matrix, double* real, double* imag) {
        auto start = std::chrono::high_resolution_clock::now();

        hipsolverHandle_t handle;
        CUSOLVER_CHECK(hipsolverDnCreate(&handle));

        // �豸�ڴ����
        double* d_matrix, * d_real, * d_imag;
        int* d_info;
        CUDA_CHECK(hipMalloc((void**)&d_matrix, 10000 * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&d_real, 100 * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&d_imag, 100 * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&d_info, sizeof(int)));

        // �������ݵ��豸
        CUDA_CHECK(hipMemcpy(d_matrix, matrix, 10000 * sizeof(double), hipMemcpyHostToDevice));

        // ���㹤���ռ�
        int lwork;
        CUSOLVER_CHECK(cusolverDnDgeev_bufferSize(
            handle,
            HIPSOLVER_EIG_MODE_NOVECTOR,
            100,
            d_matrix,
            100,
            d_real,
            d_imag,
            nullptr,
            100,
            nullptr,
            100,
            &lwork
        ));

        double* d_work;
        CUDA_CHECK(hipMalloc((void**)&d_work, lwork * sizeof(double)));

        // ��������ֵ
        CUSOLVER_CHECK(cusolverDnDgeev(
            handle,
            HIPSOLVER_EIG_MODE_NOVECTOR,
            100,
            d_matrix,
            100,
            d_real,
            d_imag,
            nullptr,
            100,
            nullptr,
            100,
            d_work,
            lwork,
            d_info
        ));

        // ���ƽ��������
        CUDA_CHECK(hipMemcpy(real, d_real, 100 * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(imag, d_imag, 100 * sizeof(double), hipMemcpyDeviceToHost));

        // ������Դ
        hipFree(d_matrix);
        hipFree(d_real);
        hipFree(d_imag);
        hipFree(d_info);
        hipFree(d_work);
        hipsolverDnDestroy(handle);

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = end - start;
        return diff.count() * 1000; // ���غ���
    }
}